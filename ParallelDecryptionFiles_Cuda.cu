#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <iostream>

using namespace std;


static const int SHIFT_VALUE = 10;
static const int NUM_OF_FILES = 6;
static const int MAX_FILE_LENGTH = 60000;
static const long DEVICE = 0;
//static const int TILE_SIZE = 512;
static const int BLK_SIZE = 512;
__constant__  int  S;
// ------------------------------------------------------------------------
//The module checks return value of the CUDA runtime call and exits the application if the call faces any errors and fails
// -------------------------------------------------------------------------

#define CUDA_CHECK_RETURN(value) {											
	hipError_t _m_cudaStat = value;										
	if (_m_cudaStat != hipSuccess) {										
		fprintf(stderr, "Error %s at line %d in file %s\n",					
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		
		exit(1);															\
	} }


// ------------------------------------------------------------------------
// the Kernel does the decryption of  caesar cipher
// dedicating task to each threads in GPU giving them unique ID as i
// -------------------------------------------------------------------------

__global__ void decrypt_caesar_cipher(char* N, char* P, int length) {
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i <= length) {
		int ch;
		ch = N[i];

		if (ch >= 'a' && ch <= 'z') {
			ch = ch - 10;

			if (ch < 'a') {
				ch = ch + 'z' - 'a' + 1;
			}

			P[i] = ch;
		}
		else if (ch >= 'A' && ch <= 'Z') {
			ch = ch - 10;

			if (ch < 'A') {
				ch = ch + 'Z' - 'A' + 1;
			}

			P[i] = ch;
		}
		P[i] = ch;
	}
}

// ------------------------------------------------------------------------
// the module open and read enccrypted files
//------------------------------------------------------------------------

int loadInputFile(string fName, char* inputArray) {
	ifstream inputFile;

	inputFile.open(fName.c_str());
	int cnt = 0;
	if (inputFile.is_open()) {
		char temp;
		while (inputFile.get(temp)) {
			inputArray[cnt++] = temp;
		}
		inputFile.close();
	}
	return cnt;
}

void writeOutput(string oName, char* output, int size) {
	ofstream outputFile;

	outputFile.open(oName.c_str());
	if (outputFile.is_open()) {
		for (size_t i = 0; i < size; i++) {
			outputFile << output[i];
		}
		outputFile.close();
	}
}

int main(void) {

	//Clear command prompt
	cout << "\033[2J\033[1;1H";
//---------------------------------------------------------------------------------------------------
//Initalize random number generator
//---------------------------------------------------------------------------------------------------
	srand(time(NULL));

	//Set device
	CUDA_CHECK_RETURN(hipSetDevice(DEVICE));
//---------------------------------------------------------------------------------------------------
//define and create dynamic array as input on host
//---------------------------------------------------------------------------------------------------
	cout << "Allocating input array on host ... ";
	int* file_lengths = new int[NUM_OF_FILES];
	char** h_N = new char* [NUM_OF_FILES];
	char** h_P = new char* [NUM_OF_FILES];
	char* d_N;
	char* d_P;
//---------------------------------------------------------------------------------------------------	
//define and create dynamic array as input on on device
//---------------------------------------------------------------------------------------------------
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_N, sizeof(char) * MAX_FILE_LENGTH));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_P, sizeof(char) * MAX_FILE_LENGTH));
//---------------------------------------------------------------------------------------------------	
// Loading 6 file decrypted files in dynamics arrays
//---------------------------------------------------------------------------------------------------
	cout << "done.\nLoading input data ... ";
	for (int i = 0; i < NUM_OF_FILES; i++) {
		h_N[i] = new char[MAX_FILE_LENGTH];
		h_P[i] = new char[MAX_FILE_LENGTH];
		int temp = loadInputFile("./encrypted" + to_string(i) + ".txt", h_N[i]);
		file_lengths[i] = temp;
	}
	cout << "done.\nCopying shift to device ... ";
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(S), &SHIFT_VALUE, sizeof(int)));
	cout << "done." << endl;


//---------------------------------------------------------------------------------------------------
//  kernel parameters 
//---------------------------------------------------------------------------------------------------
	int gridDim = ceil(MAX_FILE_LENGTH / BLK_SIZE);
	int blockDim = BLK_SIZE;


	cout << "Launching " << NUM_OF_FILES << " kernels on default stream ... ";

	hipEvent_t start, stop;
	float elapsedTime;
	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));
//---------------------------------------------------------------------------------------------------
//initiate the timer  
//---------------------------------------------------------------------------------------------------
	CUDA_CHECK_RETURN(hipEventRecord(start, 0));

//---------------------------------------------------------------------------------------------------
//Make kernel calls on default stream with defined parametres
//---------------------------------------------------------------------------------------------------
	for (int i = 0; i < NUM_OF_FILES; i++) {
		CUDA_CHECK_RETURN(hipMemcpy((void*)d_N, (void*)h_N[i], sizeof(char) * MAX_FILE_LENGTH, hipMemcpyHostToDevice));
		decrypt_caesar_cipher << < gridDim, blockDim, 0, 0 >> > (d_N, d_P, MAX_FILE_LENGTH);
		CUDA_CHECK_RETURN(hipMemcpy((void*)h_P[i], (void*)d_P, sizeof(char) * MAX_FILE_LENGTH, hipMemcpyDeviceToHost));
	}

	hipEventRecord(stop, 0);

	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipEventDestroy(start));
	CUDA_CHECK_RETURN(hipEventDestroy(stop));
	cout << "done.\nElapsed time on default stream: " << elapsedTime << " ms\n";

//---------------------------------------------------------------------------------------------------
//  Define and create the number of streams objects the same as number of files
//---------------------------------------------------------------------------------------------------
	int n_stream = NUM_OF_FILES;
	hipStream_t* ls_stream;
	ls_stream = (hipStream_t*) new hipStream_t[n_stream];
	
	for (int i = 0; i < n_stream; i++)
		hipStreamCreate(&ls_stream[i]);

	cout << "\nLaunching " << NUM_OF_FILES << " kernels with " << NUM_OF_FILES << " streams ... ";

	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));
	CUDA_CHECK_RETURN(hipEventRecord(start, 0));

//---------------------------------------------------------------------------------------------------
//stream kernels are called
//---------------------------------------------------------------------------------------------------
	// execute kernels with the CUDA stream each
	for (int i = 0; i < n_stream; i++) {
		CUDA_CHECK_RETURN(hipMemcpyAsync((void*)d_N, (void*)h_N[i], sizeof(char) * MAX_FILE_LENGTH, hipMemcpyHostToDevice, ls_stream[i]));
		decrypt_caesar_cipher << < gridDim, blockDim, 0, ls_stream[i] >> > (d_N, d_P, MAX_FILE_LENGTH);
		CUDA_CHECK_RETURN(hipMemcpyAsync((void*)h_P[i], (void*)d_P, sizeof(char) * MAX_FILE_LENGTH, hipMemcpyDeviceToHost, ls_stream[i]));
		hipStreamSynchronize(ls_stream[i]);

	}
//---------------------------------------------------------------------------------------------------
// synchronize the host and GPU
//---------------------------------------------------------------------------------------------------
	hipDeviceSynchronize();
//---------------------------------------------------------------------------------------------------
//Free stream objects memory here  ***    
//---------------------------------------------------------------------------------------------------
	for (int i = 0; i < n_stream; i++)
		hipStreamDestroy(ls_stream[i]);
	delete[] ls_stream;

//---------------------------------------------------------------------------------------------------
//stop the timer   
//---------------------------------------------------------------------------------------------------
	hipEventRecord(stop, 0);
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipEventDestroy(start));
	CUDA_CHECK_RETURN(hipEventDestroy(stop));
	cout << "done.\nElapsed time with " << NUM_OF_FILES << " kernels: " << elapsedTime << " ms\n\n";
//---------------------------------------------------------------------------------------------------
//writing out the result (decryotedfiles) from host   
//---------------------------------------------------------------------------------------------------
	cout << "Writing output ... ";
	for (int i = 0; i < NUM_OF_FILES; i++)
		writeOutput("decrypted" + to_string(i) + ".txt", h_P[i], file_lengths[i]);
//---------------------------------------------------------------------------------------------------
//free the dynamic allocation of memory from the device and host 
//---------------------------------------------------------------------------------------------------
	cout << "done.\nFreeing memory ...";
	CUDA_CHECK_RETURN(hipFree((void*)d_N));
	CUDA_CHECK_RETURN(hipFree((void*)d_P));
	CUDA_CHECK_RETURN(hipDeviceReset());

	for (int i = 0; i < NUM_OF_FILES; i++) {
		delete[] h_N[i];
		delete[] h_P[i];
	}
	delete[] h_P;
	delete[] h_N;
	delete[] file_lengths;

	cout << "done.\nExiting program\n";

	return 0;
}
